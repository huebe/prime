#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <cstring>

#define SET_BIT_CHAR_ARRAY(arr, i) ( arr[i/8] |= 1 << (i % 8) )
#define CLEAR_BIT_CHAR_ARRAY(arr, i) ( arr[i/8] &= ~(char)(1 << (i % 8)) )
#define READ_BIT_CHAR_ARRAY(arr, i) ( arr[i/8] & 1 << (i % 8) )


//WARNING, has to be (x + 1) % cMaxThreads == 0
const int cMaxThreads = 1024;
const int cMax = 2047;
const int cNumBlocks = (cMax + 1) / cMaxThreads;
const int cResults = cMax / 2 + 1;
const int cResultCharArraySize = cResults / 8;

/*
#if (((cMax + 1) % cMaxThreads ) != 0)
	#pragma error
#endif */

// memory layout: index * 2 + 1
__global__ void calculatePrimeSingle(char *isPrime) 
{
	int x = threadIdx.x;
	int y = blockIdx.x;
	int i = blockDim.x * y + x;
	int numToCheck = i * 2 + 1;
	SET_BIT_CHAR_ARRAY(isPrime, i);
	for (int j = 3; j < (numToCheck / 2) && READ_BIT_CHAR_ARRAY(isPrime, i); j += 2) {
		if ((numToCheck % j) == 0) {
			CLEAR_BIT_CHAR_ARRAY(isPrime, i);
		}
	}
}

__global__ void clearAll(char *isPrime)
{
	int x = threadIdx.x;
	int y = blockIdx.x;
	int i = blockDim.x * y + x;
	CLEAR_BIT_CHAR_ARRAY(isPrime, i);
}

__global__ void setAll(char *isPrime)
{
	int x = threadIdx.x;
	int y = blockIdx.x;
	int i = blockDim.x * y + x;
	SET_BIT_CHAR_ARRAY(isPrime, i);
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t calculatePrimeCuda(char *isPrime)
{
    char *dev_isPrime = 0;
    hipError_t cudaStatus;


    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_isPrime, cMax * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    // Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_isPrime, isPrime, cResultCharArraySize, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
	}


    // Launch a kernel on the GPU with one thread for each element.
	//calculatePrimeSingle <<<cNumBlocks, cMaxThreads >> >(dev_isPrime);

	clearAll << <cNumBlocks, cMaxThreads >> >(dev_isPrime);
	//setAll << <cNumBlocks, cMaxThreads >> >(dev_isPrime);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(isPrime, dev_isPrime, cResultCharArraySize, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_isPrime);
    
    return cudaStatus;
}


int main()
{
	char *cIsPrime = (char *)malloc(cResultCharArraySize);
	memset((void*)cIsPrime, 0x00, cResultCharArraySize);

	clock_t tStart = clock();
	hipError_t cudaStatus = calculatePrimeCuda(cIsPrime);
	clock_t tEnd = clock();

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	int numPrimeNumbers = 0;
	for (int i = 0; i < cResults; i++) {
		if (READ_BIT_CHAR_ARRAY(cIsPrime, i)) {
			printf("%i, ", i * 2 + 1);
			numPrimeNumbers++;
		}
	}

	printf("are prime numbers.\nTotal %i prime numbers.\nElapsed: %f seconds\n", numPrimeNumbers, (double)(tEnd - tStart) / CLOCKS_PER_SEC);
	free(cIsPrime);
	
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	
	getchar();

	return 0;
}